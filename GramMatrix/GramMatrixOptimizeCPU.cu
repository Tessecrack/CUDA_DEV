#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#include <limits.h>

#define PRINT_MATRIX true
#define CHECK(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);\
    } }

#define MAX_MEMORY_VECTOR 104857600 //100 Mb

#define COUNT_OF_ELEMENTS_IN_SYSTEM 1024 //Count of elements in system of vectors
#define COUNT_OF_VECTORS_IN_SYSTEM 12 //Count of vectors in system
#define COUNT_OF_ELEMENTS_IN_VECTOR (COUNT_OF_ELEMENTS_IN_SYSTEM / COUNT_OF_VECTORS_IN_SYSTEM) //Count of elements in one vector
#define SIZE_GRAM_MATRIX  (COUNT_OF_VECTORS_IN_SYSTEM * COUNT_OF_VECTORS_IN_SYSTEM)       

using namespace std;

inline void Info()
{
	cout << "Size of system: " << COUNT_OF_ELEMENTS_IN_SYSTEM 
		<< "\nCount of vectors: " << COUNT_OF_VECTORS_IN_SYSTEM
		<< "\nCount of elements in one vector: " << COUNT_OF_ELEMENTS_IN_VECTOR << endl;
}
void InfoResult(unsigned char*, unsigned char*);

void PrintSystemOfVectors(unsigned char*);

void PrintVector(unsigned char*, size_t);

unsigned char* GetRandomSystemOfVectors();

unsigned char* GetGramMatrixCPU(unsigned char* systemOfVectors, float& time);

unsigned char* GetGramMatrixGPU(unsigned char* systemOfVectors, float& time);

bool IsEqual(unsigned char* firstVector, unsigned char* secondVector, size_t size);

void Check(unsigned char* matrix_Host, unsigned char* matrix_Device);

__global__ void calculate_GramMatrix_GPU(unsigned char* systemOfVectors, unsigned char* gramMatrix)
{
	__shared__ unsigned char cache[1024];
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= SIZE_GRAM_MATRIX) return;
	for (int j = 0; j < COUNT_OF_ELEMENTS_IN_VECTOR; j++)
	{
		cache[threadIdx.x] +=
			systemOfVectors[(index / COUNT_OF_VECTORS_IN_SYSTEM) * COUNT_OF_ELEMENTS_IN_VECTOR + j] *
			systemOfVectors[(index % COUNT_OF_VECTORS_IN_SYSTEM) * COUNT_OF_ELEMENTS_IN_VECTOR + j];
	}
	__syncthreads();
	gramMatrix[index] = cache[threadIdx.x];
}
int main()
{
	Info();
	float timeCPU = 0.0f, timeGPU = 0.0f;
	unsigned char* systemOfVectors = GetRandomSystemOfVectors();
	bool isForPrint = COUNT_OF_ELEMENTS_IN_SYSTEM <= 2048;
	if (isForPrint) PrintSystemOfVectors(systemOfVectors);

	cout << "\nSize Gram matrix: " << SIZE_GRAM_MATRIX << "\n\n";

	unsigned char* matrixGramCPU = GetGramMatrixCPU(systemOfVectors, timeCPU);

	unsigned char* matrixGramGPU = GetGramMatrixGPU(systemOfVectors, timeGPU);

	Check(matrixGramCPU, matrixGramGPU);

	cout << "\n--------\n";
	cout << "Time CPU: " << timeCPU << endl;
	cout << "Time GPU: " << timeGPU << endl;
	cout << "\n--------\n";
	InfoResult(matrixGramCPU, matrixGramGPU);
	cin.get();
	return 0;
}
unsigned char* GetGramMatrixGPU(unsigned char* systemOfVectors, float& time_d)
{
	cout << "\n---------\n";
	unsigned char* matrixGram = new unsigned char[SIZE_GRAM_MATRIX];

	int memoryForGramMatrix = sizeof(unsigned char) * SIZE_GRAM_MATRIX;
	int memoryForBigVector = sizeof(unsigned char) * COUNT_OF_ELEMENTS_IN_SYSTEM;

	for (int i = 0; i < SIZE_GRAM_MATRIX; i++)
		matrixGram[i] = 0;

	unsigned char* systemOfVectors_GPU; 
	unsigned char* matrixGram_GPU;

	hipEvent_t startCUDA, stopCUDA;
	CHECK(hipEventCreate(&startCUDA));
	CHECK(hipEventCreate(&stopCUDA));

	CHECK(hipMalloc(&systemOfVectors_GPU, memoryForBigVector));
	CHECK(hipMalloc(&matrixGram_GPU, memoryForGramMatrix));

	CHECK(hipMemcpy(systemOfVectors_GPU, systemOfVectors, memoryForBigVector, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(matrixGram_GPU, matrixGram, memoryForGramMatrix, hipMemcpyHostToDevice));

	CHECK(hipEventRecord(startCUDA, 0));

	cout << "Calculate on DEVICE...\n";
	int countOfBlocks = (SIZE_GRAM_MATRIX + 1023) / 1024;
	calculate_GramMatrix_GPU<<<countOfBlocks, 1024>>>(systemOfVectors_GPU, matrixGram_GPU);
	cout << "Count of blocks: " << countOfBlocks << endl;
	

	hipEventRecord(stopCUDA, 0);
	hipEventSynchronize(stopCUDA);

	hipEventElapsedTime(&time_d, startCUDA, stopCUDA);
	time_d /= 1000;
	CHECK(hipMemcpy(matrixGram, matrixGram_GPU, memoryForGramMatrix, hipMemcpyDeviceToHost));
	cout << "Done\n";
	
	hipFree(systemOfVectors_GPU);
	hipFree(matrixGram_GPU);
	return matrixGram;
}

unsigned char* GetGramMatrixCPU(unsigned char* systemOfVectors, float& time_h)
{
	unsigned char* matrixGram = new unsigned char[SIZE_GRAM_MATRIX];
	for (int i = 0; i < SIZE_GRAM_MATRIX; i++) matrixGram[i] = 0;
	cout << "Calculate on HOST...\n";
	time_h = clock();
	for (int i = 0; i < SIZE_GRAM_MATRIX; i++)
	{
		int currentRow = (i / COUNT_OF_VECTORS_IN_SYSTEM) * COUNT_OF_VECTORS_IN_SYSTEM;
		int shiftCol  = (i / COUNT_OF_VECTORS_IN_SYSTEM);
		int currentIndexMainDiag = currentRow + shiftCol;
		if (i < currentIndexMainDiag) continue;
		unsigned char temp = 0;
		for (int j = 0; j < COUNT_OF_ELEMENTS_IN_VECTOR; j++)
			temp +=
			systemOfVectors[(i / COUNT_OF_VECTORS_IN_SYSTEM) * COUNT_OF_ELEMENTS_IN_VECTOR + j] *
			systemOfVectors[(i % COUNT_OF_VECTORS_IN_SYSTEM) * COUNT_OF_ELEMENTS_IN_VECTOR + j];
		matrixGram[currentIndexMainDiag + (i - currentIndexMainDiag) * COUNT_OF_VECTORS_IN_SYSTEM] = matrixGram[i] = temp;
	}
	cout << "Done\n";
	time_h /= CLOCKS_PER_SEC;
	return matrixGram;
}
void Check(unsigned char* matrix_Host, unsigned char* matrix_Device)
{
	cout << "\nCheck...\n";
	if (IsEqual(matrix_Host, matrix_Device, SIZE_GRAM_MATRIX))
		cout << "That's right! :)\n";
	else cout << "Wrong! :(\n";
}
bool IsEqual(unsigned char* firstVector, unsigned char* secondVector, size_t size)
{	
	for (int i = 0; i < size; i++)
		if (firstVector[i] != secondVector[i])
			return false;
	return true;
}
unsigned char* GetRandomSystemOfVectors()
{
	unsigned char* vector = new unsigned char[COUNT_OF_ELEMENTS_IN_SYSTEM];
	for (int i = 0; i < COUNT_OF_ELEMENTS_IN_SYSTEM; i++)
		vector[i] = rand() % 9 + 1;
	return vector;
}
void InfoResult(unsigned char* matrix_Host, unsigned char* matrix_Device)
{
	cout << "\nGram matrix CPU: " << endl;
	PrintVector(matrix_Host, SIZE_GRAM_MATRIX);

	cout << "\nGram matrix GPU: " << endl;
	PrintVector(matrix_Device, SIZE_GRAM_MATRIX);
}

void PrintSystemOfVectors(unsigned char* systemOfVectors)
{
	bool step = COUNT_OF_ELEMENTS_IN_SYSTEM < 10;
	cout << "\nBig vector:\n\n";
	for (int i = 0, j = 0; i < COUNT_OF_ELEMENTS_IN_SYSTEM; i++, j++)
	{
		if (j == COUNT_OF_ELEMENTS_IN_VECTOR && step)
		{
			cout << endl;
			j = 0;
		}
		cout << (int)systemOfVectors[i] << " ";
	}
	cout << endl;
}
void PrintVector(unsigned char* vector, size_t size)
{
	if (PRINT_MATRIX)
	{
		for (int i = 0; i < COUNT_OF_VECTORS_IN_SYSTEM; i++)
		{
			for (int j = 0; j < COUNT_OF_VECTORS_IN_SYSTEM; j++)
			{
				cout << (int)vector[i * COUNT_OF_VECTORS_IN_SYSTEM + j] << "\t";
			}
			cout << endl;
		}
	}
	else
	{
		for (int i = 0; i < size; i++)
			cout << (int)vector[i] << " ";
		cout << endl;
	}
}


